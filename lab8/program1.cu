#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
__global__ void addRow(int *a,int *b,int *c,int row, int col){
  int idx=threadIdx.x+blockIdx.x*blockDim.x;
  if(idx<row){
    for (int i=0;i<col;i++){
      c[idx*col+i]=a[idx*col+i]+b[idx*col+i];
    }
  }
}
__global__ void addCol(int *a,int *b,int *c,int row, int col){
  int idx=threadIdx.x+blockIdx.x*blockDim.x;
  if(idx<col){
    for (int i=0;i<row;i++){
      c[idx+row*i]=a[idx+row*i]+b[idx+row*i];
    }
  }
}
__global__ void addEach(int *a,int *b,int *c,int row, int col){
  int idx=threadIdx.x+blockIdx.x*blockDim.x;
  if(idx<row*col){
    c[idx]=a[idx]+b[idx];
  }
}
void printArray(int row,int col, int *arr){
  for(int i=0;i<row;i++){
    for(int j=0;j<col;j++){
      printf("%d ",arr[i*col+j]);
    }
    printf("\n");
  }
}
int main(){
  int m,n;
  printf("Enter the matrix size: ");
  scanf("%d",&m);
  scanf("%d",&n);
  int h_a[m*n],h_b[m*n],h_c[m*n];
  printf("Enter the matrix A element:");
  for(int i=0;i<m*n;i++)
    scanf("%d",&h_a[i]);
  printf("Enter the matrix B element: ");
  for(int i=0;i<m*n;i++)
    scanf("%d",&h_b[i]);
  int *d_a,*d_b,*d_c;
  hipMalloc(&d_a,sizeof(int)*m*n);
  hipMalloc(&d_b,sizeof(int)*m*n);
  hipMalloc(&d_c,sizeof(int)*m*n);
  hipMemcpy(d_a,h_a,sizeof(int)*m*n,hipMemcpyHostToDevice);
  hipMemcpy(d_b,h_b,sizeof(int)*m*n,hipMemcpyHostToDevice);
  addRow<<<1,m>>>(d_a,d_b,d_c,m,n);
  hipMemcpy(h_c,d_c,sizeof(int)*m*n,hipMemcpyDeviceToHost);
  printf("Row Wise Addition: \n");
  printArray(m,n,h_c);
  addCol<<<1,n>>>(d_a,d_b,d_c,m,n);
  hipMemcpy(h_c,d_c,sizeof(int)*m*n,hipMemcpyDeviceToHost);
  printf("Col Wise Addition: \n");
  printArray(m,n,h_c);
  addEach<<<m,n>>>(d_a,d_b,d_c,m,n);
  hipMemcpy(h_c,d_c,sizeof(int)*m*n,hipMemcpyDeviceToHost);
  printf("Each Addition: \n");
  printArray(m,n,h_c);

}