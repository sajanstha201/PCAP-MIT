
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
__global__ void mulRow(int *a,int *b,int *c,int row, int col){
  int idx=threadIdx.x+blockIdx.x*blockDim.x;
  if(idx<row){
    for (int i=0;i<col;i++){
      int sum=0;
      for(int k=0;k<row;k++){
          sum=sum+a[idx*col+k]*b[k*col+i];
      }
      c[idx*col+i]=sum;
    }
  }
}
__global__ void mulCol(int *a,int *b,int *c,int row, int col){
  int idx=threadIdx.x+blockIdx.x*blockDim.x;
  if(idx<col){
    for (int i=0;i<row;i++){
        int sum=0;
        for(int k=0;k<col;k++){
            sum=sum+a[k+i*col]*b[k*col+idx];
        }
        c[idx+col*i]=sum;
    }
  }
}
__global__ void mulEach(int *a,int *b,int *c,int row, int col){
  int idx=threadIdx.x+blockIdx.x*blockDim.x;
  if(idx<row*col){
    int sum=0;
    for(int k=0;k<col;k++){
      sum=sum+a[(int)(idx/col)*col+k]*b[k*col+(int)(idx%col)];
    }
    c[idx]=sum;
  }
}
void printArray(int row,int col, int *arr){
  for(int i=0;i<row;i++){
    for(int j=0;j<col;j++){
      printf("%d ",arr[i*col+j]);
    }
    printf("\n");
  }
}
int main(){
  int m,n;
  printf("Enter the matrix size: ");
  scanf("%d",&m);
  scanf("%d",&n);
  int h_a[m*n],h_b[m*n],h_c[m*n];
  printf("Enter the matrix A element:");
  for(int i=0;i<m*n;i++)
    scanf("%d",&h_a[i]);
  printf("Enter the matrix B element: ");
  for(int i=0;i<m*n;i++)
    scanf("%d",&h_b[i]);
  int *d_a,*d_b,*d_c;
  hipMalloc(&d_a,sizeof(int)*m*n);
  hipMalloc(&d_b,sizeof(int)*m*n);
  hipMalloc(&d_c,sizeof(int)*m*n);
  hipMemcpy(d_a,h_a,sizeof(int)*m*n,hipMemcpyHostToDevice);
  hipMemcpy(d_b,h_b,sizeof(int)*m*n,hipMemcpyHostToDevice);
  mulRow<<<1,m>>>(d_a,d_b,d_c,m,n);
  hipMemcpy(h_c,d_c,sizeof(int)*m*n,hipMemcpyDeviceToHost);
  printf("Row Wise mulition: \n");
  printArray(m,n,h_c);
  mulCol<<<1,n>>>(d_a,d_b,d_c,m,n);
  hipMemcpy(h_c,d_c,sizeof(int)*m*n,hipMemcpyDeviceToHost);
  printf("Col Wise mulition: \n");
  printArray(m,n,h_c);
  mulEach<<<m,n>>>(d_a,d_b,d_c,m,n);
  hipMemcpy(h_c,d_c,sizeof(int)*m*n,hipMemcpyDeviceToHost);
  printf("Each mulition: \n");
  printArray(m,n,h_c);
}