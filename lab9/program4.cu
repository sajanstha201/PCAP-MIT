#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
__global__ void op(int *a,int *b,int row, int col){
  int idx=threadIdx.x+blockIdx.x*blockDim.x;
  if(idx<row*col){
      int i=(int)(idx/col);
      int j=(int)(idx%col);
      int sum=0;
      for(int k=0;k<col;k++){
        sum=sum+a[i*col+k];
      }
      for(int k=0;k<row;k++){
        sum=sum+a[k*col+j];
      }
      b[idx]=sum;

  }
}

void printArray(int row,int col, int *arr){
  for(int i=0;i<row;i++){
    for(int j=0;j<col;j++){
      printf("%d ",arr[i*col+j]);
    }
    printf("\n");
  }
}
int main(){
  int m,n;
  printf("Enter the matrix size: ");
  scanf("%d",&m);
  scanf("%d",&n);
  int h_a[m*n],h_b[m*n];
  printf("Enter the matrix A element:");
  for(int i=0;i<m*n;i++)
    scanf("%d",&h_a[i]);
  int *d_a,*d_b;
  hipMalloc(&d_a,sizeof(int)*m*n);
  hipMalloc(&d_b,sizeof(int)*m*n);
  hipMemcpy(d_a,h_a,sizeof(int)*m*n,hipMemcpyHostToDevice);
  op<<<m,n>>>(d_a,d_b,m,n);
  hipMemcpy(h_b,d_b,sizeof(int)*m*n,hipMemcpyDeviceToHost);
  printf("Result: \n");
  printArray(m,n,h_b);
}