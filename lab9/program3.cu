#include<stdio.h>
#include<string.h>
#include<hip/hip_runtime.h>
__global__ void operation(int *arr,int m,int n){
    int id=threadIdx.x+blockDim.x*blockIdx.x;
    if(id<m*n){
      if(id%n!=0 and (id+1)%n!=0 and id>n and id<n*(m-1))
        arr[id]=~arr[id];
    }
}
int main(){
  printf("Enter number of row and col: ");
  int m,n;
  scanf("%d%d",&m,&n);
  int h_arr[m*n];
  printf("Enter the array element: ");
  for(int i=0;i<m*n;i++)
    scanf("%d",&h_arr[i]);
  int *d_arr;
  hipMalloc(&d_arr,m*n*sizeof(int));
  hipMemcpy(d_arr,h_arr,m*n*sizeof(int),hipMemcpyHostToDevice);
  operation<<<m,n>>>(d_arr,m,n);
  hipMemcpy(h_arr,d_arr,m*n*sizeof(int),hipMemcpyDeviceToHost);
  printf("Result:\n ");
  for(int i=0;i<m;i++){
    for(int j=0;j<n;j++)
      printf("%d ",h_arr[i*n+j]);
    printf("\n");
  }
}