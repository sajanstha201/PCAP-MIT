
#include<stdio.h>
#include<string.h>
#include<hip/hip_runtime.h>
__global__ void mul(int *value,int *arr,int *col_arr, int *row,int *result,int m,int n,int col){
    int id=threadIdx.x+blockDim.x*blockIdx.x;
    if(id<m){
      int sum=0;
      for(int i=row[id];i<row[id+1];i++){
          sum=sum+value[i]*arr[col_arr[i]];
      }
      result[id]=sum;
    }
}
int main(){
  printf("Enter number of row and col: ");
  int row,col;
  scanf("%d%d",&row,&col);
  int h_arr[col];
  printf("Enter the scalr vecotr of length %d: ",col);
  for(int i=0;i<col;i++)
    scanf("%d",&h_arr[i]);
  printf("Enter the number of non zero element: ");
  int n,m;
  scanf("%d",&n);
  printf("Enter the element in row array: ");
  scanf("%d",&m);
  int h_value[n],h_col[n],h_row[m];
  printf("Enter the value array:");
  for(int i=0;i<n;i++)
    scanf("%d",&h_value[i]);
  printf("Enter the col array: ");
  for(int i=0;i<n;i++)
    scanf("%d",&h_col[i]);
  printf("Enter the row array: ");
  for(int i=0;i<m;i++)
    scanf("%d",&h_row[i]);
  int *d_col,*d_value,*d_row,*d_result,*d_arr;
  hipMalloc(&d_col,n*sizeof(int));
  hipMalloc(&d_value,n*sizeof(int));
  hipMalloc(&d_result,n*sizeof(int));
  hipMalloc(&d_row,m*sizeof(int));
  hipMalloc(&d_arr,col*sizeof(int));
  hipMemcpy(d_value,h_value,n*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_col,h_col,n*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_row,h_row,m*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_arr,h_arr,col*sizeof(int),hipMemcpyHostToDevice);
  mul<<<1,n>>>(d_value,d_arr,d_col,d_row,d_result,m,n,col);
  int result[col];
  hipMemcpy(result,d_result,col*sizeof(int),hipMemcpyDeviceToHost);
  printf("Result: ");
  for(int i=0;i<col;i++)
    printf("%d ",result[i]);
}