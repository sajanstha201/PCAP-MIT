#include<stdio.h>
#include<string.h>
#include<hip/hip_runtime.h>
__global__ void operation(int *arr,int m,int n){
    int id=threadIdx.x+blockDim.x*blockIdx.x;
    if(id<m*n){
      arr[id]=pow(arr[id],(int)(arr[id]/n)+1);
    }
}
int main(){
  printf("Enter number of row and col: ");
  int m,n;
  scanf("%d%d",&m,&n);
  int h_arr[m*n];
  printf("Enter the array element: ");
  for(int i=0;i<m*n;i++)
    scanf("%d",&h_arr[i]);
  int *d_arr;
  hipMalloc(&d_arr,m*n*sizeof(int));
  hipMemcpy(d_arr,h_arr,m*n*sizeof(int),hipMemcpyHostToDevice);
  operation<<<m,n>>>(d_arr,m,n);
  hipMemcpy(h_arr,d_arr,m*n*sizeof(int),hipMemcpyDeviceToHost);
  printf("Result:\n ");
  for(int i=0;i<m;i++){
    for(int j=0;j<n;j++)
      printf("%d ",h_arr[i*n+j]);
    printf("\n");
  }
}