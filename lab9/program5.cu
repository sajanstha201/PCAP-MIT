#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
__global__ void op(int *a,char *b,char *str,int row, int col){
  int idx=threadIdx.x+blockIdx.x*blockDim.x;
  if(idx<row*col){
    int i=0;
    for(int k=0;k<idx;k++)
      i=i+a[k];
    for(int j=i;j<i+a[idx];j++)
      str[j]=b[idx];

  }
}
int main(){
  int m,n;
  printf("Enter the matrix size: ");
  scanf("%d",&m);
  scanf("%d",&n);
  int h_a[m*n];
  char h_b[m*n];
  printf("Enter the matrix int A element:");
  for(int i=0;i<m*n;i++)
    scanf("%d",&h_a[i]);
  printf("Enter the matrix char B element:");
  for(int i=0;i<m*n;i++)
    scanf(" %c",&h_b[i]);
  int *d_a;
  char *d_b;
  char *str;
  int size=0;
  for(int i=0;i<m*n;i++)
    size=size+h_a[i];
  size=size*sizeof(char);
  hipMalloc(&str,size);
  hipMalloc(&d_a,sizeof(int)*m*n);
  hipMalloc(&d_b,sizeof(char)*m*n);
  hipMemcpy(d_a,h_a,sizeof(int)*m*n,hipMemcpyHostToDevice);
  hipMemcpy(d_b,h_b,sizeof(char)*m*n,hipMemcpyHostToDevice);
  op<<<m,n>>>(d_a,d_b,str,m,n);
  char h_str[size];
  hipMemcpy(h_str,str,size,hipMemcpyDeviceToHost);
  h_str[size]='\0';
  printf("Result is %s\n",h_str);
}