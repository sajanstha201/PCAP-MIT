#include<stdio.h>
#include<string.h>
#include<hip/hip_runtime.h>
__global__ void findPattern(char *str,char *pattern, int *count,int strLen,int patternLen){
    int id=threadIdx.x+blockDim.x*blockIdx.x;
    int k=0;
    if(id<strLen-patternLen+1){
        for(int i=id;i<id+patternLen-1;i++,k++){
            if(str[i]!=pattern[k])
                return;
        }
        atomicAdd(count,1);
    }
}
int main(){
    char str[100];
    printf("Enter the sentence: ");
    fgets(str,100,stdin);
    int count=0;
    char pattern[100];
    printf("Enter a word you want to search: ");
    fgets(pattern,100,stdin);

    char *d_str;
    int *d_count;
    char *d_pattern;
    int strLen=strlen(str),patternLen=strlen(pattern);
    hipMalloc(&d_str,strLen);
    hipMalloc(&d_pattern,patternLen);
    hipMalloc(&d_count,sizeof(int));
    hipMemcpy(d_str,str,strLen,hipMemcpyHostToDevice);
    hipMemcpy(d_pattern,pattern,patternLen,hipMemcpyHostToDevice);
    hipMemcpy(d_count,&count,sizeof(int),hipMemcpyHostToDevice);
    printf("%d %d\n",strLen,patternLen);
    findPattern<<<1,strLen>>>(d_str,d_pattern,d_count,strLen,patternLen);
    hipDeviceSynchronize();
    hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);

    printf("The number of count of %s words in %s sentence is %d\n",pattern,str,count);
}