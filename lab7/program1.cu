#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<hip/hip_runtime.h>
__global__ void findPattern(char *str,char *pattern, int *count,int strLen,int patternLen){
    int id=threadIdx.x+blockdim.x*blockIdx.x;
    int k=0;
    if(id<strLen-patternLen){
        for(int i=id;i<pattenLen;i++,k++)
            if(str[i]!=pattern[k])
                return;
        atomicAdd(count,1);
    }
}
int main(){
    char str[100];
    printf("Enter the sentence: ");
    fgets(str,100,stdin);
    int count=0;
    char pattern[100];
    printf("Enter a word you want to search: ");
    fgets(pattern,100,stdin);

    char *d_str;
    int *d_count;
    char *d_pattern;
    hipMalloc(&d_str,sizeof(char)*strlen(str));
    hipMalloc(&d_pattern,sizeof(char)*strlen(pattern));
    hipMalloc(&d_count,sizeof(int));
    hipMemcpy(d_str,str,sizeof(char)*strlen(str),hipMemcpyHostToDevice);
    hipMemcpy(d_pattern,pattern,sizeof(char)*strlen(pattern),hipMemcpyHostToDevice);
    hipMemcpy(d_count,count,sizeof(int),hipMemcpyHostToDevice);
    findPattern<<<strlen(str),256>>>(d_str,d_pattern,d_count,strlen(str),strlen(pattern));
    hipMemcpy(count,d_count,sizeof(int),hipMemcpyDeviceToHost);
    printf("The number of count of %s words in %s sentence is %d\n",pattern,str,count);
}   