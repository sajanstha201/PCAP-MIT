#include<stdio.h>
#include<hip/hip_runtime.h>
#include<string.h>
__global__ void copy(char *str,char *result,int size){
    int id=threadIdx.x+blockIdx.x*blockDim.x;
    if(id<size){
        int k=0;
        for(int i=0;i<id;i++)
            k=k+(size-i-1);
        for(int i=0;i<(size-id-1);i++){
            result[k+i]=str[i];
        }
    }
}
int main(){
    char h_str[100];
    printf("Entr a string: ");
    fgets(h_str,100,stdin);
    char *d_str;
    hipMalloc(&d_str,strlen(h_str)*sizeof(char));
    int count=0;
    for(int i=1;i<=strlen(h_str);i++)
        count+=i;
    char h_result[count];
    printf("count: %d\n",count);
    char *d_result;
    hipMalloc(&d_str,strlen(h_str)*sizeof(char));
    hipMalloc(&d_result,count*sizeof(char));
    hipMemcpy(d_str,h_str,strlen(h_str)*sizeof(char),hipMemcpyHostToDevice);
    copy<<<1,strlen(h_str)>>>(d_str,d_result,strlen(h_str));
    hipMemcpy(h_result,d_result,count*sizeof(char),hipMemcpyDeviceToHost);
    printf("The result output is %s\n",h_result);
}