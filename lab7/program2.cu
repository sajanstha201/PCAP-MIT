#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<string.h>
__global__ void copy(char *str,char *result,int size){
    int id=threadIdx.x+blockIdx.x*blockDim.x;
    if(id<size){
        int k=0;
        for(int i=0;i<id;i++)
            k++;
        for(int i=0;i<id;i++)
            result[k+i]=str[i];
    }
}
int main(){
    char h_str[100];
    printf("Entr a string: ");
    fgets(h_str,100,stdin);
    char *d_str;
    hipMalloc(&d_str,sizeof(d_str)*sizeof(char));
    int count=0;
    for(int i=1;i<=strlen(d_str);i++)
        count+=i;
    char *h_result;
    char *d_str,*d_result;
    hipMalloc(&d_str,strlen(h_str)*sizeof(char));
    hipMalloc(&d_result,count*sizeof(char));
    hipMemcpy(d_str,h_str,count*sizeof(char),hipMemcpyHostToDevice);
    copy<<<1,256>>>(d_str,d_result,strlen(h_str));
    hipMemcpy(h_resutl,d_resutl,count*sizeof(char),cudamemcpyDeviceToHost);
    printf("The result output is %s\n",h_result);
}