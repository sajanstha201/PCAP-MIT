#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
__global__ void concat(char *d_str,char *d_result,int n,int l){
  int id=threadIdx.x+blockDim.x*blockIdx.x;
  if(id<n)
    for(int i=0;i<l;i++)
      d_result[id*l+i]=d_str[i];
}
int main(){
  printf("Enter a string: ");
  char h_str[100];
  fgets(h_str,100,stdin);
  int size=strlen(h_str)-1;
  char *d_str;
  hipMalloc(&d_str,size);
  hipMemcpy(d_str,h_str,size,hipMemcpyHostToDevice);
  int n;
  printf("Enter the number: ");
  scanf("%d",&n);
  char *d_result;
  hipMalloc(&d_result,n*size);

  printf("The len of stri is %d\n",size);
  concat<<<1,n>>>(d_str,d_result,n,size);
  char h_result[n*size];
  hipMemcpy(h_result,d_result,n*size,hipMemcpyDeviceToHost);
  printf("The output string is %s\n",h_result);
}