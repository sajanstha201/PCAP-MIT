#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
__global__ void reverse(char *str,int l){
  int id=threadIdx.x+blockIdx.x*blockDim.x;
  if(id<l/2){
    char temp=str[id];
    str[id]=str[l-id-1];
    str[l-id-1]=temp;
  }

}
int main(){
  char h_str[100];
  printf("Enter a string: ");
  fgets(h_str,100,stdin);
  char *d_str;
  hipMalloc(&d_str,strlen(h_str));
  hipMemcpy(d_str,h_str,strlen(h_str),hipMemcpyHostToDevice);
  reverse<<<sizeof(h_str),1>>>(d_str,strlen(h_str));
  hipMemcpy(h_str,d_str,strlen(h_str),hipMemcpyDeviceToHost);
  printf("The reverse is %s\n",h_str);

}