#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void convert(char *str,char *result,int size){
  int id=threadIdx.x+blockDim.x*blockIdx.x;
  if(id<size){
    int count=0;
    for(int i=0;i<=id;i++)
      count+=i;
    for(int i=count;i<=count+id;i++){
      result[i]=str[id];
    }
  }
}
int main(){
  char h_str[100];
  printf("Enter a string: ");
  fgets(h_str,100,stdin);
  int size=strlen(h_str)-1;
  char *d_str;
  hipMalloc(&d_str,size);
  char *d_result;
  int result_size=0;
  for(int i=0;i<=size;i++)
    result_size+=i;
  hipMalloc(&d_result,result_size);
  printf("The result size: %d\n",result_size);
  hipMemcpy(d_str,h_str,size,hipMemcpyHostToDevice);
  convert<<<1,size>>>(d_str,d_result,size);
  char result[result_size];
  hipMemcpy(result,d_result,result_size,hipMemcpyDeviceToHost);
  printf("The result is %s\n",result);
}