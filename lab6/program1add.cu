#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void convert(int *a,int *b,int n){
    int i=threadIdx.x+blockIdx.x*blockDim.x;
    if(i<n){
        int num=a[i];
        result=0;
        while(num!=0){
            result=result*10+(int)(num%8);
            num=(int)num/8;
        }
        b[i]=num;
    }
}
int main(){
    int n;
    printf("Enter the number of element: ");
    scanf("%d",&n);
    int h_a[n],h_b[n];
    printf("Enter the elements: \n");
    for(int i=0;i<n;i++)
        scanf("%d",&h_a[i]);
    int *d_a,*d_b;
    hipMalloc(&d_a,sizeof(int)*n);
    hipMalloc(&d_b ,sizeof(int)*n);
    hipMemcpy(d_a,h_a,sizeof(int)*n,hipMemcpyHostToDevice);
    convert<<<n,256>>>(d_a,d_b,n);
    hipMemcpy(h_b,d_b,sizeof(int)*n,hipMemcpyDeviceToHost);
    printf("The Octal Form are: \n")
    for(int i=0;i<n;i++)
        printf("%d ",d_b[i]);
    hipFree(d_a);
    hipFree(d_b);
}