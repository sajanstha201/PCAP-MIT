#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void conv(int *N,int *M,int *r,int n,int m){
    int id=threadIdx.x+blockIdx.x*blockDim.x;
    if(id<n){
        int value=0;
        int point=id-(int)(m/2);
        for(int i=0;i<m;i++){
            if(0<=point&&point<n){
                value+=M[i]*N[point];
            }
            point++;
        }
        r[id]=value;
    }
}
int main(){
    printf("Enter the number of element: ");
    int n;
    scanf("%d",&n);
    int N[n];
    printf("Enter the elements: ");
    for(int i=0;i<n;i++)
        scanf("%d",&N[i]);
    printf("Entr the numbee of element for Mask; ");
    int m;
    scanf("%d",&m);
    int M[m];
    printf("Enter the elements for Mark: ");
    for(int i=0;i<m;i++)
        scanf("%d",&M[i]);
    int *d_N,*d_M;
    hipMalloc(&d_N,sizeof(int)*n);
    hipMalloc(&d_M,sizeof(int)*m);
    hipMemcpy(d_N,N,sizeof(int)*n,hipMemcpyHostToDevice);
    hipMemcpy(d_M,M,sizeof(int)*m,hipMemcpyHostToDevice);
    int result[n];
    int *d_result;
    hipMalloc(&d_result,sizeof(int)*n);
    conv<<<n,256>>>(d_N,d_M,d_result,n,m);
    hipDeviceSynchronize();
    hipMemcpy(result,d_result,sizeof(int)*n,hipMemcpyDeviceToHost);
    printf("The result is :\n");
    for(int i=0;i<n;i++)
        printf("%d ",result[i]);
    hipFree(d_N);
    hipFree(d_M);
}