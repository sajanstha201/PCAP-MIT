#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void odd_even_sort(int *arr,int n,int phase){
    int id=threadIdx.x+blockDim.x*blockIdx.x;
    if(id<(int)n/2){
        int j=(phase==0)?2*id:2*id+1;
        if(j>=n||j+1>=n)
          return;
        if(arr[j]>arr[j+1]){
            int temp=arr[j+1];
            arr[j+1]=arr[j];
            arr[j]=temp;
        }
    }

}
int main(){
    int n;
    printf("Enter the number of element: ");
    scanf("%d",&n);
    int h_arr[n];
    printf("Enter the elements: ");
    for(int i=0;i<n;i++)
        scanf("%d",&h_arr[i]);
    int *d_arr;
    hipMalloc(&d_arr,sizeof(int)*n);
    hipMemcpy(d_arr,h_arr,sizeof(int)*n,hipMemcpyHostToDevice);
    for(int i=0;i<n;i++){
        odd_even_sort<<<1,n>>>(d_arr,n,i%2);
    }
    hipMemcpy(h_arr,d_arr,sizeof(int)*n,hipMemcpyDeviceToHost);
    printf("The sorted array is: ");
    for(int i=0;i<n;i++)
        printf("%d ",h_arr[i]);
}