#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
__global__ void convert(int *a,int *b,int n){
    int i=threadIdx.x+blockIdx.x*blockDim.x;
    if(i<n){
        b[i]=~a[i];
    }
}
int main(){
    printf("Enter the number of element: ");
    int n;
    scanf("%d",&n);
    printf("Enter the elements: \n");
    int h_a[n],h_b[n];
    for(int i=0;i<n;i++)
        scanf("%d",&h_a[i]);
    int *d_a,*d_b;
    hipMalloc(&d_a,sizeof(int)*n);
    hipMalloc(&d_b,sizeof(int)*n);
    hipMemcpy(d_a,h_a,sizeof(int)*n,hipMemcpyHostToDevice);
    convert<<<n,256>>>(d_a,d_b,n);
    hipMemcpy(h_b,d_b,sizeof(int)*n,hipMemcpyDeviceToHost);
    printf("The result are:\n");
    for(int i=0;i<n;i++)
        printf("%d",h_b[i]);
    

}