#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void swap(int *arr,int *index,int i){
    int temp=arr[i];
    arr[i]=arr[*index];
    arr[*index]=temp;
}
__global __void findMax(int *arr,int *max,int *index,int size){
    int id=threadIdx.x+blockIdx.x*blockDim.x;
    if(id<size){
        if(*max<arr[id]){
            max=arr[id];
            index=id;
        }
    }
}
int main(){
    int n;
    printf("Enter a number of element: ");
    scanf("%d",&n);
    int h_arr[n];
    printf("Enter the element in array: ");
    for(int i=0;i<n;i++)
        scanf("%d",&h_arr[i]);
    int *d_arr;
    hipMalloc(&d_arr,n*sizeof(int));
    hipMemcpy(d_arr,h_arr,sizeof(int)*n,hipMemcpyHostToDevice);
    for(int i=0;i<n;i++){
        int *d_max,*d_index;
        int h_max=0,h_index=0;
        hipMalloc(&d_max,sizeof(int));
        hipMalloc(&d_index,sizeof(int));
        hipMemcpy(d_max,&h_max,sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(d_index,&h_index,sizeof(int),hipMemcpyHostToDevice);
        findMax<<<1,n>>>(d_arr,d_max,d_index,n-i);
        swap<<<1,1>>>(d_arr,d_index,n-i);
    }
    hipMemcpy(h_arr,d_arr,sizeof(int)*n,hipMemcpyDeviceToHost);
    printf("The sorted aray is: ");
    for(int i=0;i<n;i++)
        printf("%d ",h_arr[i]);
}