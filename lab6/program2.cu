#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10

__global__ void selection_sort(int *d_arr, int n) {
    for (int i = 0; i < n - 1; i++) {
        int min_index = i;
        for (int j = i + 1; j < n; j++) {
            if (d_arr[j] < d_arr[min_index]) {
                min_index = j;
            }
        }
        if (min_index != i) {
            int temp = d_arr[i];
            d_arr[i] = d_arr[min_index];
            d_arr[min_index] = temp;
        }
        __syncthreads();
    }
}

void parallel_selection_sort(int *h_arr, int n) {
    int *d_arr;
    hipMalloc((void **)&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, h_arr, n * sizeof(int), hipMemcpyHostToDevice);

    selection_sort<<<1, 1>>>(d_arr, n);

    hipMemcpy(h_arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_arr);
}

int main() {
    int h_arr[N] = {64, 25, 12, 22, 11, 90, 42, 30, 8, 55};

    printf("Original Array:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    parallel_selection_sort(h_arr, N);

    printf("Sorted Array:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    return 0;
}