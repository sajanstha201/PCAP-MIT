#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) { // Ensure we don't go out of bounds
        c[idx] = a[idx] + b[idx];
        printf("a: %d b: %d index: %d c: %d\n", a[idx], b[idx], idx, c[idx]);
    }
}

int main() {
    int n;
    printf("Enter the number of elements: ");
    scanf("%d", &n);

    int h_a[n], h_b[n], h_c[n];

    printf("Enter the values for vector A: ");
    for (int i = 0; i < n; i++)
        scanf("%d", &h_a[i]);

    printf("Enter the values for vector B: ");
    for (int i = 0; i < n; i++)
        scanf("%d", &h_b[i]);

    int *d_a, *d_b, *d_c;
    int size = n * sizeof(int);

    // Fix: Use (void**) instead of (int**)
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Fix: Use a more efficient grid/block size
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    printf("The values of vector C are: ");
    for (int i = 0; i < n; i++)
        printf("%d ", h_c[i]);
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
