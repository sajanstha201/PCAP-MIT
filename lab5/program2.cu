#include "hip/hip_runtime.h"
#include<stido.h>
#include<stdlib.h>

__global__ void add(int *a,int *b,int *c,int n){
    
    int idx=threadIdx.x+blockIdx.x*blockDim.x;
    if(idx<n){
        c[idx]=a[idx]+b[idx];
    }
}
int main(){
    int n;
    printf("Enter the number of element: ");
    scanf("%d",&n);
    int h_a[n],h_b[n],h_c[n];
    printf("Enter the element of A vector: ");
    for(int i=0;i<n;i++)
        scanf("%d",&h_a[i]);
    printf("Enter the element of B vector: ");
    for(int i=0;i<n;i++)
        scanf("%d",&h_b[i]);
    int *d_a,*d_b,*d_c;
    int size=sizeof(int)*n;
    hipMalloc(&d_a,size);
    hipMalloc(&d_b,size);
    hipMalloc(&d_c,size);
    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);
    cudaMemcoy(d_b,h_b,size,hipMemcpyHostToDevice);
    int threadPerBlock=256;
    int blockNum=(int)(n/threadPerBlock+1);
    add<<<blockNum,threarPerBlock>>>(d_a,d_b,d_c,n);
    hipDeviceSynchronize();
    hipMemcpy(h_c,d_c,size,hipMemcpyDeviceToHost);
    printf("The value of C vector: ");
    for(int i;i<n;i++)
        printf("%d ",&h_c[i]);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
