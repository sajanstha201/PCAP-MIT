
#include <hip/hip_runtime.h>
#include <stdio.h>
#define MAX_ITEMS 10
#define MAX_FRIENDS 5

__global__ void calculateTotal(int *choices, float *prices, float *total, int itemsPerFriend) {
    int friendId = threadIdx.x;

    float sum = 0;
    for (int i = 0; i < itemsPerFriend; i++) {
        int itemIndex = choices[friendId * itemsPerFriend + i];
        if (itemIndex >= 0)
            sum += prices[itemIndex];
    }
    total[friendId] = sum;
}

int main() {
    int numItems, numFriends, itemsPerFriend;
    float h_prices[MAX_ITEMS];
    int h_choices[MAX_FRIENDS * MAX_ITEMS];
    float h_total[MAX_FRIENDS];

    printf("Enter number of items in menu: ");
    scanf("%d", &numItems);
    printf("Enter prices for %d items:\n", numItems);
    for (int i = 0; i < numItems; i++) {
        printf("Price of item %d: ", i);
        scanf("%f", &h_prices[i]);
    }

    printf("\nEnter number of friends: ");
    scanf("%d", &numFriends);
    printf("Enter number of items each friend will purchase: ");
    scanf("%d", &itemsPerFriend);

    for (int i = 0; i < numFriends; i++) {
        printf("\nEnter item indices (0-%d) for Friend %d:\n", numItems-1, i);
        for (int j = 0; j < itemsPerFriend; j++) {
            printf("Item %d: ", j+1);
            scanf("%d", &h_choices[i * itemsPerFriend + j]);
        }
    }

    float *d_prices, *d_total;
    int *d_choices;
    hipMalloc(&d_prices, numItems * sizeof(float));
    hipMalloc(&d_choices, numFriends * itemsPerFriend * sizeof(int));
    hipMalloc(&d_total, numFriends * sizeof(float));

    hipMemcpy(d_prices, h_prices, numItems * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_choices, h_choices, numFriends * itemsPerFriend * sizeof(int), hipMemcpyHostToDevice);

    calculateTotal<<<1, numFriends>>>(d_choices, d_prices, d_total, itemsPerFriend);
    hipMemcpy(h_total, d_total, numFriends * sizeof(float), hipMemcpyDeviceToHost);

    float finalTotal = 0;
    for (int i = 0; i < numFriends; i++) {
        printf("Friend %d total: %.2f\n", i, h_total[i]);
        finalTotal += h_total[i];
    }
    printf("Total purchase by all friends: %.2f\n", finalTotal);

    hipFree(d_prices); hipFree(d_choices); hipFree(d_total);
    return 0;
}
