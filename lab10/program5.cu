
#include <hip/hip_runtime.h>
#include <stdio.h>
#define TILE_WIDTH 8
#define MAX_WIDTH 64
#define MAX_MASK_WIDTH 5

__constant__ float d_mask[MAX_MASK_WIDTH];

__global__ void tiledConvolution1D(float *input, float *output, int width, int mask_width) {
    __shared__ float tile[TILE_WIDTH + MAX_MASK_WIDTH - 1];

    int tid = threadIdx.x;
    int start = blockIdx.x * TILE_WIDTH;
    int halo = mask_width / 2;
    int index = start + tid;

    // Load data into shared memory with halo
    if (index < width)
        tile[tid] = input[index];
    else
        tile[tid] = 0.0f;

    __syncthreads();

    if (tid < TILE_WIDTH && (start + tid) < width) {
        float result = 0.0f;
        for (int j = 0; j < mask_width; j++) {
            int dataIdx = tid + j - halo;
            if (dataIdx >= 0 && dataIdx < TILE_WIDTH + mask_width - 1)
                result += tile[dataIdx] * d_mask[j];
        }
        output[start + tid] = result;
    }
}

int main() {
    int width, mask_width;
    float h_input[MAX_WIDTH], h_output[MAX_WIDTH], h_mask[MAX_MASK_WIDTH];

    printf("Enter width of input array: ");
    scanf("%d", &width);
    printf("Enter input array elements:\n");
    for (int i = 0; i < width; i++) scanf("%f", &h_input[i]);

    printf("Enter mask width (odd number): ");
    scanf("%d", &mask_width);
    printf("Enter mask elements:\n");
    for (int i = 0; i < mask_width; i++) scanf("%f", &h_mask[i]);

    float *d_input, *d_output;
    hipMalloc(&d_input, width * sizeof(float));
    hipMalloc(&d_output, width * sizeof(float));
    hipMemcpy(d_input, h_input, width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_mask), h_mask, mask_width * sizeof(float));

    int numBlocks = (width + TILE_WIDTH - 1) / TILE_WIDTH;
    tiledConvolution1D<<<numBlocks, TILE_WIDTH + mask_width - 1>>>(d_input, d_output, width, mask_width);
    hipMemcpy(h_output, d_output, width * sizeof(float), hipMemcpyDeviceToHost);

    printf("Output after tiled 1D convolution:\n");
    for (int i = 0; i < width; i++) {
        printf("%.2f ", h_output[i]);
    }

    hipFree(d_input); hipFree(d_output);
    return 0;
}
