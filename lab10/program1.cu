#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void  mul(int *a,int *b,int *c,int n){
  int row=blockIdx.y*blockDim.y+threadIdx.y;
  int col=blockIdx.x*blockDim.x+threadIdx.x;
  if(row<n && col<n){
    int sum=0;
    for(int k=0;k<n;k++){
      sum+=a[row*n+k]*b[col+n*k];
    }
    c[row*n+col]=sum;
  }
}
int main(){
  int n;
  printf("Enter the dimension for a square matrix: ");
  scanf("%d",&n);
  int h_A[n*n],h_B[n*n],h_C[n*n];
  printf("Enter Matrix A element: ");
  for(int i=0;i<n*n;i++)
    scanf("%d",&h_A[i]);
  printf("Enter Matrix B element: ");
  for(int i=0;i<n*n;i++)
    scanf("%d",&h_B[i]);
  int *d_A,*d_B,*d_C;
  int size=n*n*sizeof(int);
  hipMalloc(&d_A,size);
  hipMalloc(&d_B,size);
  hipMalloc(&d_C,size);
  hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
  hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);
  dim3 blockD(2,2);
  dim3 gridD(int(n/blockD.x)+1,int(n/blockD.y)+1);
  mul<<<gridD,blockD>>>(d_A,d_B,d_C,n);
  hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
  printf("Result: ");
  for(int i=0;i<n;i++){
    for(int j=0;j<n;j++)
      printf("%d ",h_C[i*n+j]);
    printf("\n");
  }
}