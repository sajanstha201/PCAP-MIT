#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 5
__global__ void sc(int *a,int n){
  int id=threadIdx.x+blockDim.x*blockIdx.x;
  __shared__ int arr[N];
  if(id<n){
    arr[id]=a[id];
    __syncthreads();
    for(int i=1;i<=threadIdx.x;i=i*2){
      __syncthreads();
      arr[id]=arr[id]+arr[id-i];
    }
    a[id]=arr[id];
  }
}
int main(){
  int n=N;
  int h_a[n],h_r[n];
  printf("%d ",n);
  printf("Enter %d element for scan\n",N);
  for(int i=0;i<N;i++)
    scanf("%d",&h_a[i]);
  int *d_a,*d_r;
  int size=n*sizeof(int);
  hipMalloc(&d_a,size);
  hipMalloc(&d_r,size);
  hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);
  sc<<<1,n>>>(d_a,n);
  hipMemcpy(h_r,d_a,size,hipMemcpyDeviceToHost);
  printf("Result: ");
  for(int i=0;i<n;i++){
    printf("%d ",h_r[i]);
  }
}