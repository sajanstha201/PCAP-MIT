#include<stdio.h>
#include<hip/hip_runtime.h>
__constant__ int Mask[3];
__global__ void conv(int *N,int *r,int m,int n){
  int id=blockDim.x*blockIdx.x+threadIdx.x;
  if(id<n){
    int sum=0;
    for(int i=0;i<m;i++){
        if(!(id+i<(int)(m/2)) && !(id+i>n-(int)(m/2))){
          sum=sum+N[id+i-(int)(m/2)]*Mask[i];
        }
    }
    r[id]=sum;
  }
}
int main(){
  int n;
  printf("Enter the element: ");
  scanf("%d",&n);
  int m=3;
  int N[n],M[m],r[n];
  printf("Enter N: ");
  for(int i=0;i<n;i++)
    scanf("%d",&N[i]);
  printf("Enter 3 element of  M: ");
  for(int i=0;i<m;i++)
    scanf("%d",&M[i]);
  int *d_n,*d_r;
  hipMalloc(&d_n,n*sizeof(int));
  hipMalloc(&d_r,n*sizeof(int));
  hipMemcpy(d_n,N,n*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(Mask),M,m*sizeof(int));
  conv<<<1,n>>>(d_n,d_r,m,n);
  hipMemcpy(r,d_r,n*sizeof(int),hipMemcpyDeviceToHost);
  printf("Result: ");
  for(int i=0;i<n;i++)
    printf("%d ",r[i]);
}